#include "hip/hip_runtime.h"
/*
* This file is part of fastms.
*
* Copyright 2014 Evgeny Strekalovskiy <evgeny dot strekalovskiy at in dot tum dot de> (Technical University of Munich)
*
* fastms is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* fastms is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with fastms. If not, see <http://www.gnu.org/licenses/>.
*/

#if !defined(DISABLE_CUDA) && defined(__HIPCC__)

#include "solver_device.h"
#include "solver_base.h"
#include "util/mem_cuda.cuh"
#include "util/sum_cuda.cuh"
#include "util/timer_cuda.cuh"
#include "util/check_cuda.cuh"
#include "util/vars_cuda.cuh"



template<typename real>
class DeviceEngine: public Engine<real>
{
public:
	typedef Engine<real> Base;
	typedef typename Base::image_access_t image_access_t;
	typedef typename Base::linear_operator_t linear_operator_t;
	typedef typename Base::regularizer_t regularizer_t;
	typedef typename Base::dataterm_t dataterm_t;
	typedef DeviceAllocator allocator_t;
	typedef ImageManager<real, typename image_access_t::data_interpretation_t, allocator_t> image_manager_t;

	DeviceEngine()
	{
		summator.alloc();
		is_enabled = gpu_supports_real<real>(); if (!is_enabled) { std::cerr << "ERROR: SolverDevice<double>: Current GPU does not support double. Function calls will have no effect" << std::endl; }
	}
	virtual ~DeviceEngine()
	{
		summator.free();
	}
	virtual std::string str() { return "cuda"; }
	virtual void alloc(const ArrayDim &dim_u)
	{
		block = cuda_block_size(dim_u.w, dim_u.h);
		grid = cuda_grid_size(block, dim_u.w, dim_u.h);
	}
	virtual void free()	{}
	virtual bool is_valid() { return is_enabled; }
	virtual typename Base::image_manager_base_t* image_manager() { return &image_manager_; }
	virtual real get_sum(image_access_t a) { return summator.sum(a.const_data(), a.num_bytes()); }
	virtual void timer_start() { timer.start(); }
	virtual void timer_end() { timer.end(); }
	virtual double timer_get() { return timer.get(); }
	virtual void synchronize() { hipDeviceSynchronize(); CUDA_CHECK; }

	virtual void run_dual_p(image_access_t p, image_access_t u, linear_operator_t linear_operator, regularizer_t regularizer, real dt);
	virtual void run_prim_u(image_access_t u, image_access_t ubar, image_access_t p, linear_operator_t linear_operator, dataterm_t dataterm, real theta_bar, real dt);
	virtual void energy_base(image_access_t u, image_access_t aux_reduce, linear_operator_t linear_operator, dataterm_t dataterm, regularizer_t regularizer);
	virtual void add_edges(image_access_t cur_result, linear_operator_t linear_operator, regularizer_t regularizer);
	virtual void set_regularizer_weight_from__normgrad(image_access_t regularizer_weight, image_access_t image, linear_operator_t linear_operator);
	virtual void set_regularizer_weight_from__exp(image_access_t regularizer_weight, real coeff);
	virtual void diff_l1_base(image_access_t a, image_access_t b, image_access_t aux_reduce);

	image_manager_t image_manager_;
    DeviceTimer timer;
    DeviceSummator<real> summator;
	dim3 grid;
	dim3 block;
	bool is_enabled;
};


template<typename TImageAccess, typename TLinearOperator, typename TRegularizer>
__device__ void cuda_run_dual_p_device (const int u_num_channels, TImageAccess p, TImageAccess u, TLinearOperator linear_operator, TRegularizer regularizer, typename TImageAccess::elem_t dt)
{
	typedef typename TImageAccess::elem_t real;

	const Dim2D &dim2d = p.dim().dim2d();
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
	int x = cuda_x();
	int y = cuda_y();
	if (is_active(x, y, dim2d))
	{
		ShMemArray<real> p_sh(p_num_channels);

		linear_operator.apply(p_sh, u, x, y, dim2d, u_num_channels);

		for(int i = 0; i < p_num_channels; i++)
		{
			p_sh.get(i) = p.get(x, y, i) + p_sh.get(i) * dt;
		}

		regularizer.prox_star(p_sh, dt, x, y, dim2d, p_num_channels);

		for(int i = 0; i < p_num_channels; i++)
		{
			p.get(x, y, i) = p_sh.get(i);
		}
	}
}
template<typename TImageAccess, typename TLinearOperator, typename TRegularizer>
__global__ void cuda_run_dual_p_kernel (TImageAccess p, TImageAccess u, TLinearOperator linear_operator, TRegularizer regularizer, typename TImageAccess::elem_t dt)
{
	cuda_run_dual_p_device (u.dim().num_channels, p, u, linear_operator, regularizer, dt);
}
template<int u_num_channels, typename TImageAccess, typename TLinearOperator, typename TRegularizer>
__global__ void cuda_run_dual_p_kernel_inline (TImageAccess p, TImageAccess u, TLinearOperator linear_operator, TRegularizer regularizer, typename TImageAccess::elem_t dt)
{
	cuda_run_dual_p_device (u_num_channels, p, u, linear_operator, regularizer, dt);
}
template<typename real>
void DeviceEngine<real>::run_dual_p(image_access_t p, image_access_t u, linear_operator_t linear_operator, regularizer_t regularizer, real dt)
{
	const int u_num_channels = u.dim().num_channels;
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
	int sharedmem_p = ShMemArray<real>::size(p_num_channels, block);
	if (u_num_channels == 3)
	{
		cuda_run_dual_p_kernel_inline<3> <<<grid, block, sharedmem_p>>> (p, u, linear_operator, regularizer, dt);  CUDA_CHECK;
	}
	else
	{
		cuda_run_dual_p_kernel <<<grid, block, sharedmem_p>>> (p, u, linear_operator, regularizer, dt);  CUDA_CHECK;
	}
}


template<typename TImageAccess, typename TLinearOperator, typename TDataterm>
__device__ void cuda_run_prim_u_device (const int u_num_channels, TImageAccess u, TImageAccess ubar, TImageAccess p, TLinearOperator linear_operator, TDataterm dataterm, typename TImageAccess::elem_t theta_bar, typename TImageAccess::elem_t dt)
{
	typedef typename TImageAccess::elem_t real;

	const Dim2D &dim2d = u.dim().dim2d();
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
	int x = cuda_x();
	int y = cuda_y();
	if (is_active(x, y, dim2d))
	{
		ShMemArray<real> u_sh(u_num_channels);
		ShMemArray<real> valold_sh(u_num_channels, u_sh);

		linear_operator.apply_transpose(u_sh, p, x, y, dim2d, u_num_channels);

		for(int i = 0; i < u_num_channels; i++)
		{
			real valold = u.get(x, y, i);
			u_sh.get(i) = valold - u_sh.get(i) * dt;
			valold_sh.get(i) = valold;
		}

		dataterm.prox(u_sh, dt, x, y, dim2d, u_num_channels);

		for(int i = 0; i < u_num_channels; i++)
		{
			real valnew = u_sh.get(i);
			u.get(x, y, i) = valnew;
			real valold = valold_sh.get(i);
			ubar.get(x, y, i) = valnew + (valnew - valold) * theta_bar;
		}
	}
}
template<typename TImageAccess, typename TLinearOperator, typename TDataterm>
__global__ void cuda_run_prim_u_kernel (TImageAccess u, TImageAccess ubar, TImageAccess p, TLinearOperator linear_operator, TDataterm dataterm, typename TImageAccess::elem_t theta_bar, typename TImageAccess::elem_t dt)
{
	cuda_run_prim_u_device (u.dim().num_channels, u, ubar, p, linear_operator, dataterm, theta_bar, dt);
}
template<int u_num_channels, typename TImageAccess, typename TLinearOperator, typename TDataterm>
__global__ void cuda_run_prim_u_kernel_inline (TImageAccess u, TImageAccess ubar, TImageAccess p, TLinearOperator linear_operator, TDataterm dataterm, typename TImageAccess::elem_t theta_bar, typename TImageAccess::elem_t dt)
{
	cuda_run_prim_u_device (u_num_channels, u, ubar, p, linear_operator, dataterm, theta_bar, dt);
}
template<typename real>
void DeviceEngine<real>::run_prim_u(image_access_t u, image_access_t ubar, image_access_t p, linear_operator_t linear_operator, dataterm_t dataterm, real theta_bar, real dt)
{
	int sharedmem_2u = ShMemArray<real>::size(u.dim().num_channels, block) * 2;
	int u_num_channels = u.dim().num_channels;
	if (u_num_channels == 3)
	{
		cuda_run_prim_u_kernel_inline<3> <<<grid, block, sharedmem_2u>>> (u, ubar, p, linear_operator, dataterm, theta_bar, dt);  CUDA_CHECK;
	}
	else
	{
		cuda_run_prim_u_kernel <<<grid, block, sharedmem_2u>>> (u, ubar, p, linear_operator, dataterm, theta_bar, dt);  CUDA_CHECK;
	}
}


template<typename TImageAccess, typename TLinearOperator, typename TDataterm, typename TRegularizer>
__global__ void cuda_energy_base_kernel (TImageAccess sum, TImageAccess u, TLinearOperator linear_operator, TDataterm dataterm, TRegularizer regularizer)
{
	typedef typename TImageAccess::elem_t real;

	const Dim2D &dim2d = u.dim().dim2d();
	const int u_num_channels = u.dim().num_channels;
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
	int x = cuda_x();
	int y = cuda_y();
	if (is_active(x, y, dim2d))
	{
		real energy = real(0);

		ShMemArray<real> p_sh(p_num_channels);
		linear_operator.apply(p_sh, u, x, y, dim2d, u_num_channels);
		energy += regularizer.value(p_sh, x, y, dim2d, p_num_channels);

		__syncthreads();

		ShMemArray<real> u_sh(u_num_channels);
		for(int i = 0; i < u_num_channels; i++)
		{
			u_sh.get(i) = u.get(x, y, i);
		}
		energy += dataterm.value(u_sh, x, y, dim2d, u_num_channels);

		sum.get(x, y, 0) = energy;
	}
}
template<typename real>
void DeviceEngine<real>::energy_base(image_access_t u, image_access_t aux_reduce, linear_operator_t linear_operator, dataterm_t dataterm, regularizer_t regularizer)
{
	const int u_num_channels = u.dim().num_channels;
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
	int sharedmem_u = ShMemArray<real>::size(u_num_channels, block);
	int sharedmem_p = ShMemArray<real>::size(p_num_channels, block);
	int sharedmem_max = std::max(sharedmem_u, sharedmem_p);
	cuda_energy_base_kernel <<<grid, block, sharedmem_max>>> (aux_reduce, u, linear_operator, dataterm, regularizer); CUDA_CHECK;
}


template<typename TImageAccess, typename TLinearOperator, typename TRegularizer>
__global__ void cuda_add_edges_kernel(TImageAccess image, TLinearOperator linear_operator, TRegularizer regularizer)
{
	typedef typename TImageAccess::elem_t real;

	const Dim2D &dim2d = image.dim().dim2d();
	const int u_num_channels = image.dim().num_channels;
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
	int x = cuda_x();
	int y = cuda_y();
	if (is_active(x, y, dim2d))
	{
		ShMemArray<real> p_sh(p_num_channels);
		linear_operator.apply(p_sh, image, x, y, dim2d, u_num_channels);
	    const real max_range_norm = linear_operator.maximal_possible_range_norm(u_num_channels);
		real val_edge_indicator = regularizer.edge_indicator(p_sh, max_range_norm, x, y, dim2d, p_num_channels);
		real mult = real(1) - val_edge_indicator;
		for (int i = 0; i < u_num_channels; i++)
		{
			image.get(x, y, i) *= mult;
		}
	}
}
template<typename real>
void DeviceEngine<real>::add_edges(image_access_t cur_result, linear_operator_t linear_operator, regularizer_t regularizer)
{
	const int p_num_channels = linear_operator.num_channels_range(cur_result.dim().num_channels);
	int sharedmem_p = ShMemArray<real>::size(p_num_channels, block);
	cuda_add_edges_kernel <<<grid, block, sharedmem_p>>> (cur_result, linear_operator, regularizer); CUDA_CHECK;
}


template<typename TImageAccess, typename TLinearOperator>
__global__ void cuda_set_regularizer_weight_from__normgrad_kernel (TImageAccess regularizer_weight, TImageAccess image, TLinearOperator linear_operator)
{
	typedef typename TImageAccess::elem_t real;

	const Dim2D &dim2d = image.dim().dim2d();
	const int u_num_channels = image.dim().num_channels;
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
	int x = cuda_x();
	int y = cuda_y();
	if (is_active(x, y, dim2d))
	{
		ShMemArray<real> gradient_sh(p_num_channels);
		linear_operator.apply(gradient_sh, image, x, y, dim2d, u_num_channels);
		regularizer_weight.get(x, y, 0) = vec_norm(gradient_sh, p_num_channels);
	}
}
template<typename real>
void DeviceEngine<real>::set_regularizer_weight_from__normgrad(image_access_t regularizer_weight, image_access_t image, linear_operator_t linear_operator)
{
	const int u_num_channels = image.dim().num_channels;
    const int p_num_channels = linear_operator.num_channels_range(u_num_channels);
    int sharedmem_gradient = ShMemArray<real>::size(p_num_channels, block);
	cuda_set_regularizer_weight_from__normgrad_kernel <<<grid, block, sharedmem_gradient>>> (regularizer_weight, image, linear_operator); CUDA_CHECK;
}

template<typename TImageAccess>
__global__ void cuda_set_regularizer_weight_from__exp_kernel (TImageAccess regularizer_weight, typename TImageAccess::elem_t coeff)
{
	typedef typename TImageAccess::elem_t real;

	const Dim2D &dim2d = regularizer_weight.dim().dim2d();
	int x = cuda_x();
	int y = cuda_y();
	if (is_active(x, y, dim2d))
	{
		const real eps = real(1e-6);
		regularizer_weight.get(x, y, 0) = realmax(eps, realexp(-coeff * regularizer_weight.get(x, y, 0)));
	}
}
template<typename real>
void DeviceEngine<real>::set_regularizer_weight_from__exp(image_access_t regularizer_weight, real coeff)
{
	cuda_set_regularizer_weight_from__exp_kernel <<<grid, block>>> (regularizer_weight, coeff); CUDA_CHECK;
}


template<typename TImageAccess>
__device__ void cuda_diff_l1_base_device (const int a_num_channels, TImageAccess a, TImageAccess b, TImageAccess aux_reduce)
{
	typedef typename TImageAccess::elem_t real;

	const Dim2D &dim2d = a.dim().dim2d();
	int x = cuda_x();
	int y = cuda_y();
	if (is_active(x, y, dim2d))
	{
		real diff = real(0);
		for (int i = 0; i < a_num_channels; i++)
		{
			real val_a = a.get(x, y, i);
			real val_b = b.get(x, y, i);
			diff += realabs(val_a - val_b);
		}
		aux_reduce.get(x, y, 0) = diff;
	}
}
template<typename TImageAccess>
__global__ void cuda_diff_l1_base_kernel (TImageAccess a, TImageAccess b, TImageAccess aux_reduce)
{
	cuda_diff_l1_base_device (a.dim().num_channels, a, b, aux_reduce);
}
template<int a_num_channels, typename TImageAccess>
__global__ void cuda_diff_l1_base_kernel_inline (TImageAccess a, TImageAccess b, TImageAccess aux_reduce)
{
	cuda_diff_l1_base_device (a_num_channels, a, b, aux_reduce);
}
template<typename real>
void DeviceEngine<real>::diff_l1_base(image_access_t a, image_access_t b, image_access_t aux_reduce)
{
	const int a_num_channels = a.dim().num_channels;
	if (a_num_channels == 3)
	{
		cuda_diff_l1_base_kernel_inline<3> <<<grid, block>>> (a, b, aux_reduce);  CUDA_CHECK;
	}
	else
	{
		cuda_diff_l1_base_kernel <<<grid, block>>> (a, b, aux_reduce);  CUDA_CHECK;
	}
}




template<typename real>
class SolverDeviceImplementation: public SolverBase<real>
{
public:
	SolverDeviceImplementation() { SolverBase<real>::set_engine(&engine);	}
private:
	DeviceEngine<real> engine;
};


template<typename real> SolverDevice<real>::SolverDevice() : implementation(NULL) { implementation = new SolverDeviceImplementation<real>(); }
template<typename real> SolverDevice<real>::~SolverDevice() { delete implementation; }
template<typename real> BaseImage* SolverDevice<real>::run(const BaseImage *image, const Par &par) { return implementation->run(image, par); }
template class SolverDevice<float>;
template class SolverDevice<double>;



#endif // !defined(DISABLE_CUDA) && defined(__HIPCC__)

